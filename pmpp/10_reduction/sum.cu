#include "hip/hip_runtime.h"
#include "sum.cuh"
#include <numeric>
#include "../utils/dev_config.cuh"
#include <stdexcept>
#include <iostream>
#include "../utils/check_error.cuh"
#include "../utils/dev_vector.cuh"

namespace Numeric::CUDA
{
float sumSeq(std::vector<float> const &data)
{
    if (std::empty(data)) {
        return 0.0f;
    }
    else {
        return std::accumulate(std::begin(data), std::end(data), 0.0f);
    }
}
__global__ void sumKernelNaive(float const *data,
                               float *sum,
                               unsigned num_elems)
{
    auto const idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elems) {
        atomicAdd(sum, data[idx]);
    }
}

__global__ void sumParallelSimple(float *data, float *sum)
{
    auto const mem_loc = 2 * threadIdx.x;

    for (auto stride = 1u; stride <= blockDim.x; stride *= 2) {
        if (threadIdx.x % stride == 0) {
            data[mem_loc] += data[mem_loc + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        *sum = data[0];
    }
}

// Kernel with reduced warp divergence.
__global__ void sumParallelSimpleMinDiv(float *data, float *sum)
{
    auto const mem_loc = threadIdx.x;
    for (auto stride = blockDim.x; stride >= 1u; stride /= 2) {
        if (threadIdx.x < stride) {
            data[mem_loc] += data[mem_loc + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        *sum = data[0];
    }
}

// Kernel with reduced warp divergence and shared memory.
__global__ void sumParallelSimpleMinDivShared(float const *data, float *sum)
{
    // Copy the result of the first iteration to shared memory.
    extern __shared__ float partial_sum[];
    auto const mem_loc = threadIdx.x;
    partial_sum[mem_loc] = data[mem_loc] + data[mem_loc + blockDim.x];

    for (auto stride = blockDim.x / 2u; stride >= 1u; stride /= 2) {
        if (threadIdx.x < stride) {
            partial_sum[mem_loc] += partial_sum[mem_loc + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        *sum = partial_sum[0];
    }
}

__global__ void sumParallelSimpleMinDivSharedMultBlock(float const *data, float *sum)
{
    // Copy the result of the first iteration to shared memory.
    extern __shared__ float partial_sum[];
    auto const data_mem_loc = 2 * blockIdx.x * blockDim.x + threadIdx.x;
    auto const shared_mem_loc = threadIdx.x;
    partial_sum[shared_mem_loc] = data[data_mem_loc] + data[data_mem_loc + blockDim.x];

    for (auto stride = blockDim.x / 2u; stride >= 1u; stride /= 2) {
        __syncthreads();
        if (shared_mem_loc < stride) {
            partial_sum[shared_mem_loc] += partial_sum[shared_mem_loc + stride];
        }

    }

    if (threadIdx.x == 0) {
        atomicAdd(sum, partial_sum[0]);
    }
}

__global__ void sumParallelSimpleMinDivSharedMultBlockCoarse(float const *data, float *sum)
{
    // Copy the result of the first iteration to shared memory.
    extern __shared__ float partial_sum[];
    auto const coarse_factor = 2u;
    auto const data_mem_loc = coarse_factor * 2 * blockIdx.x * blockDim.x + threadIdx.x;

    auto first_sum = data[data_mem_loc];
    for (auto i = 1u; i < coarse_factor; ++i) {
        first_sum += data[data_mem_loc + i * blockDim.x];
    }
    auto const shared_mem_loc = threadIdx.x;
    partial_sum[shared_mem_loc] = first_sum;

    for (auto stride = blockDim.x / 2u; stride >= 1u; stride /= 2) {
        __syncthreads();
        if (shared_mem_loc < stride) {
            partial_sum[shared_mem_loc] += partial_sum[shared_mem_loc + stride];
        }

    }

    if (threadIdx.x == 0) {
        atomicAdd(sum, partial_sum[0]);
    }
}

float sumParallel(std::vector<float> const &data_host,
                  ReductionStrategy strategy)
{
    if (std::size(data_host) % 32 != 0) {
        throw std::invalid_argument{"Data size should be a multiple of 32 (warp size)\n"};
    }

    if (std::empty(data_host)) {
        return 0.0f;
    }
    else {
        auto const &dev_config = DeviceConfigSingleton::getInstance().getDevProps(0);
        auto const max_num_threads = dev_config.max_threads_per_block;
        auto const max_num_elems_valid = max_num_threads * 2u;
        if (std::size(data_host) > max_num_elems_valid) {
            throw std::invalid_argument{
                "Data size exceeds the two-time maximum number of threads per block.\n"};
        }

        auto const num_elems = std::size(data_host);

        auto const data_dev = DevVector{data_host};
        auto sum_dev = DevVector{1u, 0u};

        switch (strategy) {
            case ReductionStrategy::NAIVE: {
                auto const [grid_size, block_size] = Detail::execConfig(num_elems, strategy);
                sumKernelNaive<<<grid_size, block_size>>>(data_dev.data(),
                                                          sum_dev.data(),
                                                          num_elems);
            }
                break;

            case ReductionStrategy::SIMPLE: {
                auto const [grid_size, block_size] = Detail::execConfig(num_elems, strategy);
                sumParallelSimple<<<grid_size, block_size>>>(data_dev.data(), sum_dev.data());
            }
                break;

            case ReductionStrategy::SIMPLE_MIN_DIV: {
                auto const [grid_size, block_size] = Detail::execConfig(num_elems, strategy);
                sumParallelSimpleMinDiv<<<grid_size, block_size>>>(data_dev.data(), sum_dev.data());
            }
                break;

            case ReductionStrategy::SIMPLE_MIN_DIV_SHARED: {
                auto const [grid_size, block_size] = Detail::execConfig(num_elems, strategy);
                auto const shared_mem_size = block_size * sizeof(float);
                sumParallelSimpleMinDivShared<<<grid_size, block_size, shared_mem_size>>>(
                    data_dev.data(), sum_dev.data());
            }
                break;

            case ReductionStrategy::SIMPLE_MIN_DIV_SHARED_MULT_BLOCKS: {
                auto const [grid_size, block_size] = Detail::execConfig(num_elems, strategy);
                sumParallelSimpleMinDivSharedMultBlock<<<grid_size, block_size, block_size
                    * sizeof(float)>>>(data_dev.data(), sum_dev.data());
            }
                break;
            case ReductionStrategy::SIMPLE_MIN_DIV_SHARED_MULT_BLOCKS_COARSE: {
                auto const [grid_size, block_size] = Detail::execConfig(num_elems, strategy);
                sumParallelSimpleMinDivSharedMultBlockCoarse<<<grid_size, block_size, block_size
                    * sizeof(float)>>>(data_dev.data(), sum_dev.data());
            }
                break;
        }
        checkError(hipGetLastError(), "launch of sum kernel");

        return sum_dev.hostCopy().front();
    }
}

namespace Detail
{
std::pair<unsigned, unsigned> execConfig(unsigned num_data_elems,
                                         ReductionStrategy strategy)
{
    auto config = std::make_pair(1u, num_data_elems / 2u);

    switch (strategy) {
        case ReductionStrategy::NAIVE:
            config.second *= 2u;
            break;
        default:
            break;
    }
    return config;
}
}
}// Numeric::CUDA namespace.
