#include "hip/hip_runtime.h"
#include "vec_add.cuh"
#include "../utils/exec_config.cuh"
#include <stdexcept>
#include "../utils/check_error.cuh"
#include "../utils/dev_timer.cuh"
#include "../utils/dev_vector.cuh"
#include "hip/hip_runtime_api.h"
#include "../utils/dev_vector_factory.cuh"
#include "../utils/host_dev_copy.cuh"

namespace PMPP::CUDA
{
__global__ void vecAddKernel(float const *first, float const *sec,
                             float *res, unsigned num_elems);

__global__ void vecAddKernel(float const *first, float const *sec,
                             float *res, unsigned num_elems)
{
    auto const idx{blockIdx.x * blockDim.x + threadIdx.x};
    if (idx < num_elems) {
        res[idx] = first[idx] + sec[idx];
    }
}

std::vector<float> vecAdd(std::vector<float> const &first_host,
                          std::vector<float> const &sec_host,
                          bool print_kernel_time)
{
    if (std::size(first_host) != std::size(sec_host)) {
        throw std::invalid_argument{"Vector size should be equal\n"};
    }

    // Allocate device vectors and transfer input data.
    auto first_dev = DevVectorFactory::create(first_host);
    auto sec_dev = DevVectorFactory::create(sec_host);

    // Allocate result vector in the device.
    auto const num_elems{static_cast<unsigned>(std::size(first_host))};
    auto res_dev{DevVector<float>{num_elems}};

    // Execute the kernel.
    auto const exec_params{ExecConfig::getParams(num_elems, vecAddKernel, 0u)};
    auto timer{DevTimer{}};
    timer.tic();

    //hipProfilerStart();
    vecAddKernel<<<exec_params.grid_dim, exec_params.block_dim>>>(
        first_dev.data(), sec_dev.data(), res_dev.data(), num_elems);
    checkErrorKernel("vector addition kernel", true);
    //hipProfilerStop();
    auto time_taken_sec{timer.toc()};

    if (print_kernel_time) {
        std::cout << "Time taken (kernel:vec_add):: " << time_taken_sec * 1000.0 << " milli seconds." << std::endl;
    }

    return HostDevCopy::hostCopy(res_dev);
}

void vecAddExample()
{
    auto const dev_prop = DeviceConfigSingleton::getInstance().getDevProps(0);
    auto const max_num_thread_sm = dev_prop.max_threads_per_sm;
    auto const num_sm = dev_prop.num_sm;

    auto const num_elems = max_num_thread_sm * num_sm * 1000u;
    auto const first_host = std::vector<float>(num_elems, 1.0f);
    auto const sec_host = std::vector<float>(num_elems, 2.0f);
    vecAdd(first_host, sec_host, true);
}

}// Numeric namespace.

