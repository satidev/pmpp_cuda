#include "dev_config.cuh"
#include <iostream>

DeviceConfigSingleton::DeviceConfigSingleton()
{
    auto num_dev = 0;
    hipGetDeviceCount(&num_dev);
    num_dev_ = static_cast<unsigned>(num_dev);
    setDevProperties();
}

void DeviceConfigSingleton::setDevProperties()
{
    dev_props_.reserve(num_dev_);

    for (auto dev_id = 0; dev_id < num_dev_; dev_id++) {

        auto dev_prop = hipDeviceProp_t{};
        hipGetDeviceProperties(&dev_prop, dev_id);
        auto dev_prop_struct = DeviceProperties{};

        dev_prop_struct.device_id = dev_id;
        dev_prop_struct.num_sm = dev_prop.multiProcessorCount;
        dev_prop_struct.global_mem_size = dev_prop.totalGlobalMem;
        dev_prop_struct.constant_mem_size = dev_prop.totalConstMem;
        dev_prop_struct.max_threads_per_sm = dev_prop.maxThreadsPerMultiProcessor;
        dev_prop_struct.max_threads_per_block = dev_prop.maxThreadsPerBlock;
        dev_prop_struct.max_shared_mem_per_sm = dev_prop.sharedMemPerMultiprocessor;
        dev_prop_struct.max_shared_mem_per_block = dev_prop.sharedMemPerBlock;
        dev_prop_struct.max_regs_per_block = dev_prop.regsPerBlock;
        dev_prop_struct.max_regs_per_sm = dev_prop.regsPerMultiprocessor;
        dev_prop_struct.max_threads_per_warp = dev_prop.warpSize;
        dev_prop_struct.max_blocks_per_sm = dev_prop.maxBlocksPerMultiProcessor;
        dev_prop_struct.warp_size = dev_prop.warpSize;
        dev_props_.push_back(dev_prop_struct);
    }
}

void DeviceConfigSingleton::printDeviceProperties(unsigned dev_id) const
{
    if(dev_id >= num_dev_) {
        throw std::invalid_argument{"Invalid device ID."};
    }

    auto const dev_prop = dev_props_[dev_id];
    std::cout << "Device ID: " << dev_prop.device_id << std::endl;
    std::cout << "Number of SMs: " << dev_prop.num_sm << std::endl;
    std::cout << "Global memory size (bytes): " << dev_prop.global_mem_size << std::endl;
    std::cout << "Constant memory size (bytes): " << dev_prop.constant_mem_size << std::endl;
    std::cout << "Max threads per SM: " << dev_prop.max_threads_per_sm << std::endl;
    std::cout << "Max threads per block: " << dev_prop.max_threads_per_block << std::endl;
    std::cout << "Max shared memory per SM: " << dev_prop.max_shared_mem_per_sm << std::endl;
    std::cout << "Max shared memory per block: " << dev_prop.max_shared_mem_per_block << std::endl;
    std::cout << "Max registers per block: " << dev_prop.max_regs_per_block << std::endl;
    std::cout << "Max registers per SM: " << dev_prop.max_regs_per_sm << std::endl;
    std::cout << "Max threads per warp: " << dev_prop.max_threads_per_warp << std::endl;
    std::cout << "Max blocks per SM: " << dev_prop.max_blocks_per_sm << std::endl;
    std::cout << "Warp size: " << dev_prop.warp_size << std::endl;
}

