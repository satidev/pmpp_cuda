#include "hip/hip_runtime.h"
#include "copy_execute_latency.cuh"
#include "../../utils/dev_timer.cuh"
#include "../../utils/check_error.cuh"
#include "../../utils/exec_config.cuh"

namespace BPNV::CopyExecuteLatency
{
__global__ void sqKernel(float const *ip, float *op,
                         unsigned num_elems)
{
    auto const idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elems) {
        op[idx] = ip[idx] * ip[idx];
    }
}

MilliSeconds seqCopyExecutePageable(unsigned num_elems)
{
    // Allocate input data in host memory.
    auto constexpr init_val = 2.0f;
    auto const input_data_host = std::vector<float>(num_elems, init_val);

    // Allocate input data in device memory and transfer the data.
    auto const num_bytes = num_elems * sizeof(float);
    auto input_dat_dev = static_cast<float *>(nullptr);
    checkError(hipMalloc(reinterpret_cast<void **>(&input_dat_dev), num_bytes),
               "allocating device memory for input data");

    // Allocate device memory for the result.
    auto output_data_dev = static_cast<float *>(nullptr);
    checkError(hipMalloc(reinterpret_cast<void **>(&output_data_dev), num_bytes),
               "allocating device memory for output data");

    // Allocate the host memory for the result.
    auto res_host = std::vector<float>(num_elems);
    auto const exec_params = ExecConfig::getParams(num_elems, sqKernel, 0u);

    hipDeviceSynchronize();
    auto timer = DevTimer{};
    timer.tic();

    // Copy the input data to the device.
    checkError(hipMemcpy(input_dat_dev,
                          input_data_host.data(),
                          num_bytes,
                          hipMemcpyHostToDevice),
               "copying data to device");
    // Execute the kernel.
    sqKernel<<<exec_params.grid_dim, exec_params.block_dim>>>(
        input_dat_dev, output_data_dev, num_elems);

    // Copy the result from the device to the host.
    checkError(hipMemcpy(res_host.data(),
                          output_data_dev,
                          num_bytes,
                          hipMemcpyDeviceToHost),
               "copying data to host");

    hipDeviceSynchronize();
    auto const duration = timer.toc();
    hipFree(input_dat_dev);
    hipFree(output_data_dev);

    if (!Detail::hasSameVal(res_host, init_val * init_val)) {
        std::cerr << "Error: Kernel execution failed\n";
        std::exit(1);
    }

    return duration;
}

MilliSeconds seqCopyExecuteUnified(unsigned num_elems)
{
    auto constexpr init_val = 2.0f;

    // Allocate input data in unified memory that can be
    // accessed by both the host and the device.
    auto const num_bytes = num_elems * sizeof(float);
    auto input_data = static_cast<float *>(nullptr);
    checkError(hipMallocManaged(reinterpret_cast<void **>(&input_data), num_bytes),
               "allocating unified memory for input data");
    for (auto i = 0u; i < num_elems; ++i) {
        input_data[i] = init_val;
    }

    // Allocate unified memory for the result.
    auto output_data = static_cast<float *>(nullptr);
    checkError(hipMallocManaged(reinterpret_cast<void **>(&output_data), num_bytes),
               "allocating device memory for output data");
    auto const exec_params = ExecConfig::getParams(num_elems, sqKernel, 0u);
    // No explicit data transfer is required.
    // Execute the kernel.
    hipDeviceSynchronize();
    auto timer = DevTimer{};
    timer.tic();
    sqKernel<<<exec_params.grid_dim, exec_params.block_dim>>>(
        input_data, output_data, num_elems);
    hipDeviceSynchronize();
    auto const duration = timer.toc();

    if (!Detail::hasSameVal(std::span < float > {output_data, num_elems}, init_val * init_val)) {
        std::cerr << "Error: Kernel execution failed\n";
        std::exit(1);
    }

    // Clean up.
    hipFree(input_data);
    hipFree(output_data);

    return duration;
}

MilliSeconds seqCopyExecutePinned(unsigned num_elems)
{
    // Allocate input data in host memory.
    auto constexpr init_val = 2.0f;
    auto const input_data_host = std::vector<float>(num_elems, init_val);
    checkError(hipHostRegister((void *) input_data_host.data(), num_elems * sizeof(float),
                                hipHostRegisterDefault),
               "registering input data host memory");

    // Allocate input data in device memory and transfer the data.
    auto const num_bytes = num_elems * sizeof(float);
    auto input_data_dev = static_cast<float *>(nullptr);
    checkError(hipMalloc(reinterpret_cast<void **>(&input_data_dev), num_bytes),
               "allocating device memory for input data");

    // Allocate device memory for the result.
    auto output_data_dev = static_cast<float *>(nullptr);
    checkError(hipMalloc(reinterpret_cast<void **>(&output_data_dev), num_bytes),
               "allocating device memory for output data");

    // Allocate the host memory for the result.
    auto res_host = std::vector<float>(num_elems);
    checkError(hipHostRegister((void *) res_host.data(), num_elems * sizeof(float),
                                hipHostRegisterDefault),
               "registering result host memory");
    auto const exec_params = ExecConfig::getParams(num_elems, sqKernel, 0u);

    hipDeviceSynchronize();
    auto timer = DevTimer{};
    timer.tic();

    // Copy the input data to the device.
    checkError(hipMemcpyAsync(input_data_dev,
                               input_data_host.data(),
                               num_bytes,
                               hipMemcpyHostToDevice),
               "copying data to device");
    // Execute the kernel.
    sqKernel<<<exec_params.grid_dim, exec_params.block_dim>>>(
        input_data_dev, output_data_dev, num_elems);

    // Copy the result from the device to the host.
    checkError(hipMemcpyAsync(res_host.data(),
                               output_data_dev,
                               num_bytes,
                               hipMemcpyDeviceToHost),
               "copying data to host");

    hipDeviceSynchronize();
    auto const duration = timer.toc();

    // Clean up.
    hipFree(input_data_dev);
    hipFree(output_data_dev);
    checkError(hipHostUnregister((void *) input_data_host.data()),
               "unregistering input data host memory");
    checkError(hipHostUnregister((void *) res_host.data()),
               "unregistering input data host memory");

    if (!Detail::hasSameVal(res_host, init_val * init_val)) {
        std::cerr << "Error: Kernel execution failed\n";
        std::exit(1);
    }

    return duration;
}

MilliSeconds stagedConcurrentCopyExecute(unsigned num_elems, unsigned num_streams)
{
    // Allocate input data in host memory.
    auto constexpr init_val = 2.0f;
    auto const input_data_host = std::vector<float>(num_elems, init_val);
    checkError(hipHostRegister((void *) input_data_host.data(), num_elems * sizeof(float),
                                hipHostRegisterDefault),
               "registering input data host memory");

    // Allocate input data in device memory and transfer the data.
    auto const num_elem_stream = num_elems / num_streams;
    auto const num_byte_stream = num_elem_stream * sizeof(float);
    auto input_dat_dev = static_cast<float *>(nullptr);
    checkError(hipMalloc(reinterpret_cast<void **>(&input_dat_dev), num_byte_stream),
               "allocating device memory for input data");

    // Allocate device memory for the result.
    auto output_data_dev = static_cast<float *>(nullptr);
    checkError(hipMalloc(reinterpret_cast<void **>(&output_data_dev), num_byte_stream),
               "allocating device memory for output data");

    // Allocate the host memory for the result.
    auto res_host = std::vector<float>(num_elems);
    checkError(hipHostRegister((void *) res_host.data(), num_elems * sizeof(float),
                                hipHostRegisterDefault),
               "registering result host memory");

    // Allocate streams.
    auto streams = std::vector<hipStream_t>(num_streams);
    for (auto &stream: streams) {
        checkError(hipStreamCreate(&stream), "creating stream");
    }

    auto const exec_params = ExecConfig::getParams(num_elem_stream,
                                                   sqKernel, 0u);

    hipDeviceSynchronize();

    auto timer = DevTimer{};
    timer.tic();
    for (auto i = 0u; i < num_streams; ++i) {
        // Copy the input data to the device.
        auto const offset = i * num_elem_stream;
        checkError(hipMemcpyAsync(input_dat_dev,
                                   input_data_host.data() + offset,
                                   num_byte_stream,
                                   hipMemcpyHostToDevice,
                                   streams[i]),
                   "copying data to device");
        // Execute the kernel.
        sqKernel<<<exec_params.grid_dim, exec_params.block_dim, 0, streams[i]>>>(
            input_dat_dev, output_data_dev, num_elem_stream);
        // Copy the result from the device to the host.
        checkError(hipMemcpyAsync(res_host.data() + offset,
                                   output_data_dev,
                                   num_byte_stream,
                                   hipMemcpyDeviceToHost,
                                   streams[i]),
                   "copying data to host");
    }

    hipDeviceSynchronize();
    auto const duration = timer.toc();

    // Clean up.
    for (auto &stream: streams) {
        hipStreamDestroy(stream);
    }
    hipFree(input_dat_dev);
    hipFree(output_data_dev);
    checkError(hipHostUnregister((void *) input_data_host.data()),
               "unregistering input data host memory");
    checkError(hipHostUnregister((void *) res_host.data()),
               "unregistering input data host memory");

    if (!Detail::hasSameVal(res_host, init_val * init_val)) {
        std::cerr << "Error: Kernel execution failed\n";
        std::exit(1);
    }

    return duration;
}

MilliSeconds zeroCopyExecute(unsigned num_elems)
{
    // Allocate input data in host memory.
    auto constexpr init_val = 2.0f;
    auto const input_data_host = std::vector<float>(num_elems, init_val);

    // Check the device properties.
    auto dev_prop = hipDeviceProp_t{};
    checkError(hipGetDeviceProperties(&dev_prop, 0), "getting device properties");
    if (!dev_prop.canMapHostMemory) {
        std::cerr << "Error: Zero copy memory is not supported\n";
        std::exit(1);
    }

    checkError(hipSetDeviceFlags(hipDeviceMapHost),
               "setting device flags for zero copy memory");
    checkError(hipHostRegister((void *) input_data_host.data(), num_elems * sizeof(float),
                                hipHostRegisterMapped),
               "registering input data host memory");
    auto input_data_mapped = static_cast<float *>(nullptr);
    checkError(hipHostGetDevicePointer((void **) &input_data_mapped,
                                        (void *) input_data_host.data(), 0),
               "getting device pointer for mapped memory");

    // Allocate the host memory for the result.
    auto res_dev = static_cast<float *>(nullptr);
    checkError(hipMallocManaged((void **) &res_dev, num_elems * sizeof(float)),
               "allocating device memory for output data");

    hipDeviceSynchronize();
    auto timer = DevTimer{};
    timer.tic();

    auto const exec_params = ExecConfig::getParams(num_elems, sqKernel, 0u);
    sqKernel<<<exec_params.grid_dim, exec_params.block_dim>>>(
        input_data_mapped, res_dev, num_elems);

    hipDeviceSynchronize();
    auto const duration = timer.toc();


    if (!Detail::hasSameVal(std::span < float > {res_dev, num_elems}, init_val * init_val)) {
        std::cerr << "Error: Kernel execution failed\n";
        std::exit(1);
    }

    // Clean up.
    // Deregister the host memory.
    checkError(hipHostUnregister((void *) input_data_host.data()),
               "unregistering input data host memory");
    hipFree(res_dev);


    return duration;
}

PerfTestResult runPerfTest(unsigned num_rep)
{
    auto constexpr num_elems = 1u << 24;
    auto perf_info = PerfTestResult{};

    for (auto i = 0u; i < num_rep; ++i) {
        perf_info["seq-pageable"].emplace_back(seqCopyExecutePageable(num_elems).count());
        perf_info["seq-unified"].emplace_back(seqCopyExecuteUnified(num_elems).count());
        perf_info["seq-pinned"].emplace_back(seqCopyExecutePinned(num_elems).count());
        perf_info["seq-zero-copy"].emplace_back(zeroCopyExecute(num_elems).count());
        perf_info["staged-concurrent"].emplace_back(
            stagedConcurrentCopyExecute(num_elems, 256u).count());
    }
    return perf_info;
}

namespace Detail
{
bool hasSameVal(std::span<float> vec, float val)
{
    // Check if all the elements in the vector have the same value.
    return std::all_of(std::begin(vec), std::end(vec),
                       [val](float elem)
                       { return elem == val; });
}
}// Detail namespace.
}// Latency namespace.
