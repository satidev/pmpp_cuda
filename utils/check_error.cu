#include "check_error.cuh"
#include <stdexcept>

void checkError(hipError_t result, std::string const &func_desc)
{
    if (result != hipSuccess)
    {
        throw std::runtime_error{
            "CUDA runtime error during " + func_desc + ".\n" + hipGetErrorString(result)};
    }
}
void checkErrorKernel(std::string const &kern_desc, bool block)
{
    // Checking synchronous errors like invalid execution configuration.
    auto const err_sync = hipGetLastError();
    if (err_sync != hipSuccess)
    {
        throw std::runtime_error{
            "CUDA kernel (synchronous) error during " + kern_desc + ".\n" + hipGetErrorString(err_sync)};
    }

    if (block)
    {
        // Checking asynchronous errors like out-of-bound memory within the kernel.
        auto const err_async = hipDeviceSynchronize();
        if (err_async != hipSuccess)
        {
            throw std::runtime_error{
                "CUDA kernel (asynchronous) error during " + kern_desc + ".\n" + hipGetErrorString(err_sync)};
        }
    }
}
