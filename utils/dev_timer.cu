#include "dev_timer.cuh"
#include "check_error.cuh"

DevTimer::DevTimer()
{
    checkError(hipEventCreate(&start_), "creation of start event of timer");
    checkError(hipEventCreate(&stop_), "creation of stop event of timer");
}

DevTimer::~DevTimer()
{
    checkError(hipEventDestroy(start_), "destruction of start event of timer");
    checkError(hipEventDestroy(stop_), "destruction of stop event of timer");
}
void DevTimer::tic(hipStream_t stream)
{
    stream_ = stream;
    checkError(hipEventRecord(start_, stream_), "hipEventRecord");
}

MilliSeconds DevTimer::toc()
{
    checkError(hipEventRecord(stop_, stream_), "hipEventRecord for timer stop function");
    checkError(hipEventSynchronize(stop_), "hipEventSynchronize for timer stop function");

    auto elapsed_time_ms = 0.0f;
    checkError(hipEventElapsedTime(&elapsed_time_ms, start_, stop_),
               "timer's elapsed time computation");
    return MilliSeconds{elapsed_time_ms};
}

