#include "hip/hip_runtime.h"
#include "sum.cuh"
#include <numeric>
#include "../utils/dev_config.cuh"
#include <stdexcept>
#include <iostream>
#include "../utils/check_error.cuh"

namespace Numeric::CUDA
{
float sumSeq(std::vector<float> const &data)
{
    if (std::empty(data)) {
        return 0.0f;
    }
    else {
        return std::accumulate(std::begin(data), std::end(data), 0.0f);
    }
}
// Similar to the sequential addition.
__global__ void sumKernelNaive(float const *data,
                               float *sum,
                               unsigned num_elems)
{
    auto const idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elems) {
        atomicAdd(sum, data[idx]);
    }
}

__global__ void sumParallelSimple(float *data,
                                  float *sum,
                                  unsigned num_elems)
{
    auto const mem_loc = 2 * threadIdx.x;

    for (auto stride = 1u; stride < num_elems; stride *= 2) {
        if (threadIdx.x % stride == 0) {
            data[mem_loc] += data[mem_loc + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        *sum = data[0];
    }
}

// Kernel with reduced warp divergence.
__global__ void sumParallelSimpleMinDiv(float *data,
                                        float *sum,
                                        unsigned num_elems)
{
    auto const mem_loc = threadIdx.x;
    for (auto stride = blockDim.x; stride >= 1u; stride /= 2) {
        if (threadIdx.x < stride) {
            data[mem_loc] += data[mem_loc + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        *sum = data[0];
    }
}

// Kernel with reduced warp divergence and shared memory.
__global__ void sumParallelSimpleMinDivShared(float const *data,
                                              float *sum,
                                              unsigned num_elems)
{
    // Copy the result of the first iteration to shared memory.
    extern __shared__ float partial_sum[];
    auto const mem_loc = threadIdx.x;
    partial_sum[mem_loc] = data[mem_loc] + data[mem_loc + blockDim.x];

    for (auto stride = blockDim.x/2u; stride >= 1u; stride /= 2) {
        if (threadIdx.x < stride) {
            partial_sum[mem_loc] += partial_sum[mem_loc + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        *sum = partial_sum[0];
    }
}

float sumParallel(std::vector<float> const &data_host)
{
    if (std::size(data_host) % 32 != 0) {
        throw std::invalid_argument{"Data size should be a multiple of 32 (warp size)\n"};
    }

    if (std::empty(data_host)) {
        return 0.0f;
    }
    else {
        auto const &dev_config = DeviceConfigSingleton::getInstance().getDevProps(0);
        auto const max_num_threads = dev_config.max_threads_per_block;
        auto const max_num_elems_valid = max_num_threads * 2u;
        if (std::size(data_host) > max_num_elems_valid) {
            throw std::invalid_argument{
                "Data size exceeds the two-time maximum number of threads per block.\n"};
        }

        auto const num_elems = std::size(data_host);
        auto const data_size_bytes = num_elems * sizeof(float);

        auto data_dev = static_cast<float *>(nullptr);
        checkError(hipMalloc(reinterpret_cast<void **>(&data_dev), data_size_bytes),
                   "allocation of device buffer for data");
        checkError(hipMemcpy(data_dev,
                              std::data(data_host),
                              data_size_bytes,
                              hipMemcpyHostToDevice),
                   "transfer of data from the host to the device");

        // Currently, only a single block can be launched.
        // It is better to keep the number of threads per block as a multiple of warp size.
        auto const block_size = static_cast<unsigned>(
            std::ceil(static_cast<float>(num_elems) / 64.0f)) * 32u;
        if (block_size > max_num_threads) {
            throw std::invalid_argument{
                "Block size exceeds the maximum number of threads per block.\n"};
        }
        auto const grid_size = 1u;
        auto sum_dev = static_cast<float *>(nullptr);
        hipMalloc(reinterpret_cast<void **>(&sum_dev), sizeof(float));
        hipMemset(sum_dev, 0, sizeof(float));

        sumParallelSimpleMinDivShared<<<grid_size, block_size>>>(data_dev, sum_dev, num_elems);
        checkError(hipGetLastError(), "launch of sumKernel");

        auto sum_host = 0.0f;
        checkError(hipMemcpy(&sum_host, sum_dev, sizeof(float), hipMemcpyDeviceToHost),
                   "transfer of sum from the device to the host");

        hipFree(data_dev);
        hipFree(sum_dev);

        return sum_host;
    }
}

}// Numeric::CUDA namespace.
