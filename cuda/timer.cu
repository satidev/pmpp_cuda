#include "timer.cuh"
#include "check_error.cuh"


Timer::Timer()
{
    checkError(hipEventCreate(&start_), "creation of start event of timer");
    checkError(hipEventCreate(&stop_), "creation of stop event of timer");
}

Timer::~Timer()
{
    checkError(hipEventDestroy(start_), "destruction of start event of timer");
    checkError(hipEventDestroy(stop_), "destruction of stop event of timer");
}
void Timer::tic()
{
    checkError(hipEventRecord(start_), "hipEventRecord");
}

float Timer::toc()
{
    checkError(hipEventRecord(stop_), "hipEventRecord for timer stop function");
    checkError(hipEventSynchronize(stop_), "hipEventSynchronize for timer stop function");

    auto elapsed_time_ms = 0.0f;
    checkError(hipEventElapsedTime(&elapsed_time_ms, start_, stop_),
               "timer's elapsed time computation");
    return elapsed_time_ms/1000.0f;
}

