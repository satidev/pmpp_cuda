#include "hip/hip_runtime.h"
#include "histogram.cuh"
#include "check_error.cuh"
#include "exec_config.cuh"

namespace Numeric::CUDA
{
__global__ void hist_kern(unsigned short const *data,
                          unsigned *hist,
                          unsigned num_data_elems)
{
    auto const idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < num_data_elems){

        if(data[idx] == 0){
            atomicAdd(&(hist[0]), 1u);
        }
        else{
            atomicAdd(&(hist[1]), 1u);
        }
    }
}

std::vector<unsigned> histogram(std::vector<bool> const &data_host)
{
    if (std::empty(data_host)) {
        return std::vector<unsigned>{};
    }
    // Convert input data to unsigned shorts.
    auto const data_i_host = std::vector<unsigned short>(std::begin(data_host), std::end(data_host));

    auto const num_data_elems = std::size(data_i_host);
    auto const num_hist_bins = 2u;

    // Histogram output in host.
    auto hist_host = std::vector<unsigned>(num_hist_bins);

    auto const ip_size_bytes = num_data_elems * sizeof(unsigned short);
    auto const op_size_bytes = num_hist_bins * sizeof(unsigned);

    // Copy input data to the device.
    auto data_i_dev = static_cast<unsigned short *>(nullptr);
    auto hist_dev = static_cast<unsigned *>(nullptr);

    checkError(hipMalloc(reinterpret_cast<void **>(&data_i_dev), ip_size_bytes),
               "allocation of device buffer for input vector");
    checkError(hipMalloc(reinterpret_cast<void **>(&hist_dev), op_size_bytes),
               "allocation of device buffer for histogram output");

    // Transfer data from the host to the device.
    checkError(hipMemcpy(data_i_dev, std::data(data_i_host), ip_size_bytes, hipMemcpyHostToDevice),
               "transfer of data from the input vector to the device");

    // Initialize histogram buffer to zero.
    checkError(hipMemset(hist_dev, 0u, op_size_bytes),
               "Initialize histogram buffer");
    // Execute the kernel.
    // The Number of threads launched depended on the number of elements in the input vector.
    auto const exec_params = ExecConfig::getParams(num_data_elems, hist_kern, 0u);
    hist_kern<<<exec_params.grid_dim, exec_params.block_dim>>>(
        data_i_dev, hist_dev, static_cast<unsigned>(num_data_elems));

    checkErrorKernel("Histogram kernel", true);


    // Transfer result data from the device to host.
    checkError(hipMemcpy(std::data(hist_host), hist_dev, op_size_bytes, hipMemcpyDeviceToHost),
               "transfer results from the device to host");

    hipFree(data_i_dev);
    hipFree(hist_dev);

    return hist_host;
}

}// Numeric::CUDA namespace.
