#include "hip/hip_runtime.h"
#include "conv_2d.cuh"
#include "../cpp/eigen_utils.h"

namespace Numeric::CUDA
{
__global__ void conv_kern_2d(float const *data,
                             float const *filter,
                             float *res,
                             unsigned num_rows,
                             unsigned num_cols,
                             unsigned filter_radius);


__global__ void conv_kern_2d(float const *data,
                             float const *filter,
                             float *res,
                             unsigned num_rows,
                             unsigned num_cols,
                             unsigned filter_radius)
{
    auto const row = blockIdx.y * blockDim.y + threadIdx.y;
    auto const col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < num_rows && col < num_cols) {

        auto const filter_size = 2u * filter_radius + 1u;

        // Element by element multiplication and accumulation.
        auto sum = 0.0f;

        for (auto filt_row = 0u; filt_row < filter_size; ++filt_row) {
            for (auto filt_col = 0u; filt_col < filter_size; ++filt_col) {
                // Flipped indices.
                auto const data_row = static_cast<int>(row) - static_cast<int>(filt_row) +
                                    static_cast<int>(filter_radius);
                auto const data_col = static_cast<int>(col) - static_cast<int>(filt_col) +
                                    static_cast<int>(filter_radius);
                if (data_row >= 0 && data_row < num_rows && data_col >= 0 && data_col < num_cols) {
                    sum += data[data_row * num_cols + data_col] * filter[filt_row * filter_size + filt_col];
                }
            }
        }
        res[row * num_cols + col] = sum;
    }

}


Eigen::MatrixXf conv2D(Eigen::MatrixXf const &data,
                       Eigen::MatrixXf const &filter,
                       bool use_shared_mem)
{
    // Check the data size.
    if (data.size() == 0u) {
        throw std::invalid_argument("Data size is zero.");
    }

    // Check the filter size.
    if (filter.size() == 0u) {
        throw std::invalid_argument("Filter size is zero.");
    }

    // Make sure that the 2D filter is square.
    if (filter.rows() != filter.cols()) {
        throw std::invalid_argument("Filter is not square.");
    }

    // Check the filter size is odd.
    if (filter.rows() % 2u == 0u) {
        throw std::invalid_argument("Filter size is not odd.");
    }

    // Copy Eigen matrix objects to STL vector to copy to GPU.
    auto const data_vec = EigenUtils::toVec(data);
    auto const filter_vec = EigenUtils::toVec(filter);
    auto res_vec = std::vector<float>(data.size(), 0.0f);

    // Copy the data and filter to the GPU.
    auto const data_size_bytes = data.size() * sizeof(float);
    auto const filter_size_bytes = filter.size() * sizeof(float);
    auto const res_size_bytes = res_vec.size() * sizeof(float);

    auto d_data = static_cast<float*>(nullptr);
    auto d_filter = static_cast<float*>(nullptr);
    auto d_res = static_cast<float*>(nullptr);
    hipMalloc(reinterpret_cast<void**>(&d_data), data_size_bytes);
    hipMalloc(reinterpret_cast<void**>(&d_filter), filter_size_bytes);
    hipMalloc(reinterpret_cast<void**>(&d_res), res_size_bytes);

    // Transfer data from the host to the device.
    hipMemcpy(d_data, data_vec.data(), data_size_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_filter, filter_vec.data(), filter_size_bytes, hipMemcpyHostToDevice);

    // Launch the kernel.
    if(use_shared_mem) {

    }
    else {
        auto const block_size = dim3{16u, 16u};
        auto const num_block_x = (static_cast<unsigned>(data.cols()) + block_size.x - 1u) / block_size.x;
        auto const num_block_y = (static_cast<unsigned>(data.rows()) + block_size.y - 1u) / block_size.y;
        auto const grid_size = dim3{num_block_x, num_block_y};

        auto const filter_radius = static_cast<unsigned>(filter.rows() / 2u);
        conv_kern_2d<<<grid_size, block_size>>>(d_data, d_filter, d_res,
                                                static_cast<unsigned>(data.rows()),
                                                static_cast<unsigned>(data.cols()),
                                                filter_radius);
    }
    // Transfer the result back to the host.
    hipMemcpy(res_vec.data(), d_res, res_size_bytes, hipMemcpyDeviceToHost);

    return EigenUtils::toMat<float>(res_vec, data.rows(), data.cols());
}
}// Numeric::CUDA namespace.
