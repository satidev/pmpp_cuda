#include "hip/hip_runtime.h"
#include "mat_mul.cuh"

namespace Numeric::CUDA
{
__global__ void mat_mul(float *a, float *b, float *res,
                        unsigned num_rows_a, unsigned num_cols_a,
                        unsigned num_cols_b)
{
    auto const row = blockIdx.y * blockDim.y + threadIdx.y;
    auto const col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < num_rows_a && col < num_cols_b) {
        auto res_elem_val = 0.0f;
        for (auto idx = 0u; idx < num_cols_a; idx++) {
            res_elem_val += (a[row * num_cols_a + idx] * b[idx * num_cols_b + col]);
        }
        res[row * num_cols_b + col] = res_elem_val;
    }
}

auto constexpr TILE_WIDTH = 16u;

__global__ void mat_mul_square_sm(float *a, float *b, float *res,
                                  unsigned num_cols)
{
    auto const row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    auto const col = blockIdx.x * TILE_WIDTH + threadIdx.x;

    __shared__ float a_tile[TILE_WIDTH][TILE_WIDTH];
    __shared__ float b_tile[TILE_WIDTH][TILE_WIDTH];

    auto const num_rows = num_cols;
    auto const num_tiles = static_cast<unsigned>(ceil(static_cast<float>(num_rows) / TILE_WIDTH));
    for (auto tile_idx = 0u; tile_idx < num_tiles; tile_idx++) {
        // Load the tiles into shared memory.
        auto const a_row = row;
        auto const b_col = col;
        auto const a_col = tile_idx * TILE_WIDTH + threadIdx.x;
        auto const b_row = tile_idx * TILE_WIDTH + threadIdx.y;

        if (a_row < num_rows && a_col < num_cols) {
            a_tile[threadIdx.y][threadIdx.x] = a[a_row * num_cols + a_col];
        }
        else {
            a_tile[threadIdx.y][threadIdx.x] = 0.0f;
        }
        if (b_row < num_rows && b_col < num_cols) {
            b_tile[threadIdx.y][threadIdx.x] = b[b_row * num_cols + b_col];
        }
        else {
            b_tile[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        // Compute the result for the tiles.
        auto res_elem_val = 0.0f;
        for (auto idx = 0u; idx < TILE_WIDTH; idx++) {
            res_elem_val += (a_tile[threadIdx.y][idx] * b_tile[idx][threadIdx.x]);
        }

        // Store the result in the global memory.
        if (row < num_rows && col < num_cols) {
            res[row * num_cols + col] += res_elem_val;
        }
        __syncthreads();

    }
}


Eigen::MatrixXf matMul(Eigen::MatrixXf const &a, Eigen::MatrixXf const &b)
{
    if (a.cols() != b.rows()) {
        throw std::invalid_argument{"Invalid size for matrix multiplication."};
    }

    // Copy Eigen matrix objects to STL vector to copy to GPU.
    auto const a_vec = EigenUtils::toVec(a);
    auto const b_vec = EigenUtils::toVec(b);
    auto res_vec = std::vector<float>(a.rows() * b.cols(), -1.0f);

    // Allocate device GPU memory.
    auto a_vec_dev = static_cast<float *>(nullptr);
    auto b_vec_dev = static_cast<float *>(nullptr);
    auto res_vec_dev = static_cast<float *>(nullptr);
    hipMalloc(reinterpret_cast<void **>(&a_vec_dev), a_vec.size() * sizeof(float));
    hipMalloc(reinterpret_cast<void **>(&b_vec_dev), b_vec.size() * sizeof(float));
    hipMalloc(reinterpret_cast<void **>(&res_vec_dev), res_vec.size() * sizeof(float));

    // Transfer input matrix elements to GPU.
    hipMemcpy(a_vec_dev, a_vec.data(), a_vec.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_vec_dev, b_vec.data(), b_vec.size() * sizeof(float), hipMemcpyHostToDevice);

    // Execute the kernel.
    auto const num_threads_per_block = dim3{16u, 16u};
    auto const num_blocks_x = static_cast<unsigned>(
        std::ceil(static_cast<float>(b.cols()) /
            static_cast<float>(num_threads_per_block.x)));
    auto const num_blocks_y = static_cast<unsigned>(
        std::ceil(static_cast<float>(a.rows()) /
            static_cast<float>(num_threads_per_block.y)));
    auto const num_blocks = dim3{num_blocks_x, num_blocks_y};
    mat_mul<<<num_blocks, num_threads_per_block>>>(a_vec_dev,
                                                   b_vec_dev,
                                                   res_vec_dev,
                                                   static_cast<unsigned>(a.rows()),
                                                   static_cast<unsigned>(a.cols()),
                                                   static_cast<unsigned>(b.cols()));

    hipMemcpy(res_vec.data(), res_vec_dev, res_vec.size() * sizeof(float), hipMemcpyDeviceToHost);

    return EigenUtils::toMat<float>(res_vec, a.rows(), b.cols());
}



Eigen::MatrixXf matMulSharedMem(Eigen::MatrixXf const &a,
                                Eigen::MatrixXf const &b)
{
    if ((a.rows() != a.cols()) || (b.rows() != b.cols())) {
        throw std::invalid_argument{
            "Currently matrix multiplication is supported only for square matrices."};
    }
    if ((a.rows() != b.rows() || (a.cols() != b.cols()))) {
        throw std::invalid_argument{"Invalid size for matrix multiplication."};
    }

    // Copy Eigen matrix objects to STL vector to copy to GPU.
    auto const a_vec = EigenUtils::toVec(a);
    auto const b_vec = EigenUtils::toVec(b);
    auto res_vec = std::vector<float>(a_vec.size());
    auto const vec_size_bytes = a_vec.size() * sizeof(float);

    // Allocate device GPU memory.
    auto a_vec_dev = static_cast<float *>(nullptr);
    auto b_vec_dev = static_cast<float *>(nullptr);
    auto res_vec_dev = static_cast<float *>(nullptr);
    hipMalloc(reinterpret_cast<void **>(&a_vec_dev), vec_size_bytes);
    hipMalloc(reinterpret_cast<void **>(&b_vec_dev), vec_size_bytes);
    hipMalloc(reinterpret_cast<void **>(&res_vec_dev), vec_size_bytes);

    // Transfer input matrix elements to GPU.
    hipMemcpy(a_vec_dev, a_vec.data(), vec_size_bytes, hipMemcpyHostToDevice);
    hipMemcpy(b_vec_dev, b_vec.data(), vec_size_bytes, hipMemcpyHostToDevice);

    // Execute the kernel.
    auto const num_threads_per_block = dim3{TILE_WIDTH, TILE_WIDTH};
    auto const num_blocks_x = static_cast<unsigned>(
        std::ceil(static_cast<float>(a.rows()) /
            static_cast<float>(num_threads_per_block.x)));
    auto const num_blocks = dim3{num_blocks_x, num_blocks_x};
    mat_mul_square_sm<<<num_blocks, num_threads_per_block>>>(a_vec_dev,
                                                             b_vec_dev,
                                                             res_vec_dev,
                                                             static_cast<unsigned>(a.rows()));

    hipMemcpy(res_vec.data(), res_vec_dev, vec_size_bytes, hipMemcpyDeviceToHost);

    return EigenUtils::toMat<float>(res_vec, a.rows(), a.cols());
}

} //Numeric namespace.