#include "hip/hip_runtime.h"
#include "stencil_1d.cuh"
#include "exec_config.cuh"
#include "check_error.cuh"

namespace Numeric::CUDA
{

__global__ void diff_kernel(float const *ip, float *op,
                            unsigned num_elems_ip)
{
    auto const idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= 1 && idx < num_elems_ip) {
        op[idx - 1] = ip[idx] - ip[idx - 1];
    }
}

__global__ void sum_3point_kernel(float const *ip, float *op,
                                  unsigned num_elems)
{
    auto const idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= 1u && idx < (num_elems - 1)) {
        op[idx] = ip[idx - 1] + ip[idx] + ip[idx + 1];
    }
    else if (idx == 0u || idx == (num_elems - 1)) {
        op[idx] = ip[idx];
    }
}

std::vector<float> diff(std::vector<float> const &ip_vec)
{
    // There should be a minimum 2 elements in the input vector.
    if (std::size(ip_vec) < 2u) {
        throw std::invalid_argument{"Input vector should have at least 2 elements."};
    }

    auto const num_elems_diff = std::size(ip_vec) - 1u;
    auto diff_vec = std::vector<float>(num_elems_diff);

    auto const ip_vec_size_bytes = ip_vec.size() * sizeof(float);
    auto const diff_vec_size_bytes = num_elems_diff * sizeof(float);

    // Allocate device memory.
    auto ip_vec_dev = static_cast<float *>(nullptr);
    auto diff_vec_dev = static_cast<float *>(nullptr);

    checkError(hipMalloc(reinterpret_cast<void **>(&ip_vec_dev), ip_vec_size_bytes),
               "allocation of device buffer for input vector");
    checkError(hipMalloc(reinterpret_cast<void **>(&diff_vec_dev), diff_vec_size_bytes),
               "allocation of device buffer for diff vector");

    // Transfer data from the host to the device.
    checkError(hipMemcpy(ip_vec_dev, ip_vec.data(), ip_vec_size_bytes, hipMemcpyHostToDevice),
               "transfer of data from the input vector to the device");

    // Execute the kernel.
    // The Number of threads launched depended on the number of elements in the input vector.
    auto const exec_params = ExecConfig::getParams(ip_vec.size(), diff_kernel, 0u);
    diff_kernel<<<exec_params.grid_dim, exec_params.block_dim>>>(
        ip_vec_dev, diff_vec_dev, static_cast<unsigned >(ip_vec.size()));

    checkErrorKernel("Diff kernel", true);

    // Transfer result data from the device to host.
    checkError(hipMemcpy(diff_vec.data(),
                          diff_vec_dev,
                          diff_vec_size_bytes,
                          hipMemcpyDeviceToHost),
               "transfer results from the device to host");

    hipFree(ip_vec_dev);
    hipFree(diff_vec_dev);

    return diff_vec;
}

std::vector<float> sum3Point(std::vector<float> const &ip_vec,
                             bool use_shared_mem)
{
    // There should be a minimum 3 elements in the input vector.
    if (std::size(ip_vec) < 3u) {
        throw std::invalid_argument{"Input vector should have at least 3 elements."};
    }

    auto const num_elems = std::size(ip_vec);
    auto sum_vec = std::vector<float>(num_elems, 0.0f);

    auto const vec_size_bytes = num_elems * sizeof(float);

    // Allocate device memory.
    auto ip_vec_dev = static_cast<float *>(nullptr);
    auto sum_vec_dev = static_cast<float *>(nullptr);

    checkError(hipMalloc(reinterpret_cast<void **>(&ip_vec_dev), vec_size_bytes),
               "allocation of device buffer for input vector");
    checkError(hipMalloc(reinterpret_cast<void **>(&sum_vec_dev), vec_size_bytes),
               "allocation of device buffer for sum vector");

    // Transfer data from the host to the device.
    checkError(hipMemcpy(ip_vec_dev, ip_vec.data(), vec_size_bytes, hipMemcpyHostToDevice),
               "transfer of data from the input vector to the device");

    // Execute the kernel.
    if(use_shared_mem)
    {

    }
    else
    {
        // The Number of threads launched depended on the number of elements in the input vector.
        auto const exec_params = ExecConfig::getParams(ip_vec.size(), sum_3point_kernel, 0u);
        sum_3point_kernel<<<exec_params.grid_dim, exec_params.block_dim>>>(
            ip_vec_dev, sum_vec_dev, static_cast<unsigned >(ip_vec.size()));
    }

    checkErrorKernel("sum 3-point kernel", true);

    // Transfer result data from the device to host.
    checkError(hipMemcpy(sum_vec.data(),
                          sum_vec_dev,
                          vec_size_bytes,
                          hipMemcpyDeviceToHost),
               "transfer results from the device to host");

    hipFree(ip_vec_dev);
    hipFree(sum_vec_dev);

    return sum_vec;
}

}// Numeric::CUDA namespace.
